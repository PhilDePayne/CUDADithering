#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "BmpUtil.h"
#include "Common.h"

#define	f7_16	112	
#define	f5_16	 80	
#define	f3_16	 48	
#define	f1_16	 16	

byte imgRes[256 * 256 * 4];
byte imgRes2[256 * 256 * 4];
const int width = 256;
const int height = 256;
int imageW, imageH;

hipError_t makeDitherThreshold(const byte* h_Src, int imageW, int imageH, int bytesPerPixel);
hipError_t makeDitherFSRgbNbpp(const byte* h_Src, int imageW, int imageH, int bytesPerPixel);

__global__ void test(byte* a)
{
    for (int i = 0; i < 256; i++) {
        for (int j = 0; j < 256; j++) {
            for (int k = 0; k < 4; k++) {

                if ((a[i * 256 * 4 + j * 4] + a[i * 256 * 4 + j * 4 + 1] + a[i * 256 * 4 + j * 4 + 2]) / 3 < 127) {
                    a[i * 256 * 4 + j * 4 + 0] = 0;
                    a[i * 256 * 4 + j * 4 + 1] = 0;
                    a[i * 256 * 4 + j * 4 + 2] = 0;
                    a[i * 256 * 4 + j * 4 + 3] = 255;
                }

                else {
                    a[i * 256 * 4 + j * 4 + 0] = 255;
                    a[i * 256 * 4 + j * 4 + 1] = 255;
                    a[i * 256 * 4 + j * 4 + 2] = 255;
                    a[i * 256 * 4 + j * 4 + 3] = 255;
                }

            }
        }
    }

}

__global__ void fsd2(byte* a)
{
    const int size = 256 * 256;

    int* error = (int*)malloc(size * sizeof(int));

    memset(error, 0, size * sizeof(int));

    int	i = 0;

    for (int y = 0; y < height; y++)
    {
        byte* prow = a + (y * width * 4);

        for (int x = 0; x < width; x++, i++)
        {
            const int	blue = prow[x * 4 + 0];
            const int	green = prow[x * 4 + 1];
            const int	red = prow[x * 4 + 2];

            //	Get the pixel gray value.
            int	newVal = (red + green + blue) / 3 + (error[i] >> 8);	//	PixelGray + error correction

            int	newc = (newVal < 128 ? 0 : 255);
            prow[x * 4 + 0] = newc;	//	blue
            prow[x * 4 + 1] = newc;	//	green
            prow[x * 4 + 2] = newc;	//	red

            //	Correction - the new error
            const int	cerror = newVal - newc;

            int idx = i + 1;
            if (x + 1 < width)
                error[idx] += (cerror * f7_16);

            idx += width - 2;
            if (x - 1 > 0 && y + 1 < height)
                error[idx] += (cerror * f3_16);

            idx++;
            if (y + 1 < height)
                error[idx] += (cerror * f5_16);

            idx++;
            if (x + 1 < width && y + 1 < height)
                error[idx] += (cerror * f1_16);
        }
    }

    free(error);
}

int main()
{
    char imageName[] = "peppers.bmp";

    byte* pixels;
    int32 width;
    int32 height;
    int32 bytesPerPixel;
    ReadImage(imageName, &pixels, &width, &height, &bytesPerPixel);

    hipError_t status = makeDitherThreshold(pixels, width, height, bytesPerPixel);

    status = makeDitherFSRgbNbpp(pixels, width, height, bytesPerPixel);

    free(pixels);

    return 0;
}

hipError_t makeDitherThreshold(const byte* h_Src, int imageW, int imageH, int bytesPerPixel) {

    byte* imgArray = 0;
    byte resultLocal[256 * 256 * 4];

    hipError_t error;

    error = hipMalloc((void**)&imgArray, imageW * imageH * 4 * sizeof(byte));
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! %s\n", hipGetErrorString(error));
    }

    error = hipMemcpy(imgArray, h_Src, imageW * imageH * 4 * sizeof(byte), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        error = hipGetLastError();
        fprintf(stderr, "hipMemcpy failed! %s\n", hipGetErrorString(error));
    }

    fsd2 << <1, 1 >> > (imgArray);

    // Check for any errors launching the kernel
    error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(error));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", error);
    }

    error = hipMemcpy(imgRes, imgArray, imageW * imageH * 4 * sizeof(byte), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMemcpy Device2Host failed: %s\n", hipGetErrorString(error));
    }


    char resultName[] = "output2.bmp";

    WriteImage(resultName, imgRes2, imageW, imageH, bytesPerPixel);

    hipFree(imgArray);
    return error;

}

hipError_t makeDitherFSRgbNbpp(const byte* h_Src, int imageW, int imageH, int bytesPerPixel) {

    byte* imgArray = 0;
    byte resultLocal[256 * 256 * 4];

    hipError_t error;

    error = hipMalloc((void**)&imgArray, imageW * imageH * 4 * sizeof(byte));
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! %s\n", hipGetErrorString(error));
    }

    error = hipMemcpy(imgArray, h_Src, imageW * imageH * 4 * sizeof(byte), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        error = hipGetLastError();
        fprintf(stderr, "hipMemcpy failed! %s\n", hipGetErrorString(error));
    }

    fsd2 << <1, 1 >> > (imgArray);

    // Check for any errors launching the kernel
    error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(error));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", error);
    }

    error = hipMemcpy(imgRes, imgArray, imageW * imageH * 4 * sizeof(byte), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMemcpy Device2Host failed: %s\n", hipGetErrorString(error));
    }


    char resultName[] = "output1.bmp";

    WriteImage(resultName, imgRes, imageW, imageH, bytesPerPixel);

    char resultName2[] = "output2.bmp";

    error = hipMalloc((void**)&imgArray, imageW * imageH * 4 * sizeof(byte));
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! %s\n", hipGetErrorString(error));
    }

    error = hipMemcpy(imgArray, h_Src, imageW * imageH * 4 * sizeof(byte), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        error = hipGetLastError();
        fprintf(stderr, "hipMemcpy failed! %s\n", hipGetErrorString(error));
    }

    test << <1, 1 >> > (imgArray);

    // Check for any errors launching the kernel
    error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(error));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", error);
    }

    error = hipMemcpy(imgRes, imgArray, imageW * imageH * 4 * sizeof(byte), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMemcpy Device2Host failed: %s\n", hipGetErrorString(error));
    }

    WriteImage(resultName2, imgRes, imageW, imageH, bytesPerPixel);

    hipFree(imgArray);
    return error;

};


